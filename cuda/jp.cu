#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <omp.h>

#include "graph.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// NOTE: This code currently infinite-loops if the number of colors is greater than COLOR_MASK_SIZE.
// These values are chosen due to the limited number of registers available in CUDA
#define COLOR_MASK_SIZE 500
#define BLOCK_SIZE 128

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if(abort) {
            exit(code);
        }
    }
}
#else
#define cudaCheckError(ans) ans
#endif

// Stores all data structures allocated on CUDA device
struct cudaContext {
    int numVertices;
    int numEdges;
    int *weights;   // Length numVertices     - List of weights for vertices
    int *vertices;  // Length numVertices+1   - List of graph vertices, in CSR representation
    int *neighbors; // Length 2*numEdges      - List of vertex neighbors, in CSR representation
    int *worklist;  // Length numVertices     - Boolean array indicating whether each vertex is in worklist
    int *coloring;  // Length numVertices     - Integer array indicating color of each vertex
    int *worklistEmptyFlag;   // Boolean flag indicating whether the worklist is empty
    int *worklistChangedFlag; // Boolean flag indicating whether the worklist was changed
};

/**
 * @brief Set all of the random states to be used for hiprand
 * @param[in] context All data structures allocated on CUDA device
 * @param[in] seed The seed to be used for creating the hiprand states
 * @param[out] states The random states created by the kernel function
 */
__global__ void kernelRandInit(struct cudaContext context, unsigned int seed, hiprandState_t *states) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (0 <= v && v < context.numVertices) {
        hiprand_init(seed, v, 0, &states[v]);
    }
}

/**
 * @brief Set all vertex weights to be random numbers, which will allow us to find independent sets
 * @param[in] context All data structures allocated on CUDA device
 * @param[in] states The hiprand states to initialize random numbers
 */
__global__ void kernelSetWeights(struct cudaContext context, hiprandState_t *states) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    
    int numVertices = context.numVertices;
    if (0 <= v && v < numVertices && context.worklist[v]) {
        context.weights[v] = hiprand(&states[v]) % numVertices;
    }
}

/**
 * @brief Creates an independent set of vertices and colors them, and removes colored vertices from worklist
 * @param[in] context All data structures allocated on CUDA device
 * @param[in] color The color to fill in the independent set with
 */
__global__ void kernelColorJP(struct cudaContext context, int color) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int numVertices = context.numVertices;
    const int *vertices = context.vertices;
    const int *neighbors = context.neighbors;
    int *weights = context.weights;
    int *worklist = context.worklist;
    int *coloring = context.coloring;
    int *worklistEmptyFlag = context.worklistEmptyFlag;
    int *worklistChangedFlag = context.worklistChangedFlag;

    if(v < numVertices && worklist[v]) {

        // Check neighbors and determine the remaining worklist
        for(int i = vertices[v]; i < vertices[v+1]; i++) {
            int w = neighbors[i];
            if(weights[v] <= weights[w] && v != w) {
                // Indicate that worklist is not empty and elements still remain
                *worklistEmptyFlag = 0;
                return;
            }
        }
        *worklistChangedFlag = 1;
        coloring[v] = color;
        worklist[v] = 0;
    }
}

/**
 * @brief Copy graph representation to CUDA device and allocate all necessary data structures
 * @param[in] graph The graph to color
 * @return context A cudaContext struct containing all device data structures
 */
struct cudaContext setup(const Graph &graph) {
    int numVertices = graph.getNumVertices();
    int numEdges = graph.getNumEdges();
    const int *vertices = graph.getCSRVertices();
    const int *neighbors = graph.getCSRNeighbors();

    // Initialize data structures
    struct cudaContext context;
    context.numVertices = numVertices;
    context.numEdges = numEdges;
    hipMalloc(&context.weights, sizeof(int) * numVertices);
    hipMalloc(&context.vertices, sizeof(int) * (numVertices + 1));
    hipMalloc(&context.neighbors, sizeof(int) * 2 * numEdges);
    hipMalloc(&context.worklist, sizeof(int) * numVertices);
    hipMalloc(&context.coloring, sizeof(int) * numVertices);
    hipMalloc(&context.worklistEmptyFlag, sizeof(int));
    hipMalloc(&context.worklistChangedFlag, sizeof(int));
   
    // Note that hipMemset sets values per byte, so 0x01 => 0x01010101 = 16843009
    hipMemcpy(context.vertices, vertices, sizeof(int) * (numVertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(context.neighbors, neighbors, sizeof(int) * 2 * numEdges, hipMemcpyHostToDevice);
    hipMemset(context.worklist, 0x01, sizeof(int) * numVertices);
    hipMemset(context.weights, 0x00, sizeof(int) * numVertices);
    hipMemset(context.coloring, 0x00, sizeof(int) * numVertices);
    hipMemset(context.worklistEmptyFlag, 0x00, sizeof(int));
    hipMemset(context.worklistChangedFlag, 0x00, sizeof(int));

    return context;
}

/**
 * @brief Frees all data structures in the given CUDA context
 * @param[in] context The CUDA context to free
 */
void freeCudaContext(struct cudaContext context) {
    hipFree(context.weights);
    hipFree(context.vertices);
    hipFree(context.neighbors);
    hipFree(context.worklist);
    hipFree(context.coloring);
    hipFree(context.worklistEmptyFlag);
    hipFree(context.worklistChangedFlag);
}

/**
 * @brief Performs graph coloring using the JP algorithm
 * @param[in] context A cudaContext sturct containing all device data structures
 * @return coloring The outputted coloring
 */
const int *jpColoring(struct cudaContext context) {
    int numVertices = context.numVertices;

    // Define kernel size and device flags
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((numVertices + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int worklistEmptyFlag = 0;
    int worklistChangedFlag = 0;
    int color = 0;

    // Initialize random states
    hiprandState_t *states;
    hipMalloc((void**)&states, sizeof(hiprandState_t) * numVertices);
    kernelRandInit<<<gridDim, blockDim>>>(context, time(NULL), states);
    hipDeviceSynchronize();

    // Loop until worklist is empty
    while(!worklistEmptyFlag) {
        // Set random vertex weights for each vertex
        hipMemset(context.weights, 0x00, sizeof(int) * numVertices);
        kernelSetWeights<<<gridDim, blockDim>>>(context, states);
        hipDeviceSynchronize();
 
        // Resolve conflicts and determine the remaining worklist
        hipMemset(context.worklistEmptyFlag, 0x01, sizeof(int));
        hipMemset(context.worklistChangedFlag, 0x00, sizeof(int));
        kernelColorJP<<<gridDim, blockDim>>>(context, color);
        hipDeviceSynchronize();
        hipMemcpy(&worklistEmptyFlag, context.worklistEmptyFlag, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&worklistChangedFlag, context.worklistChangedFlag, sizeof(int), hipMemcpyDeviceToHost);

        if (worklistChangedFlag) color++;
    }
    // Cleanup
    hipFree(states);

    // Retrieve coloring from device
    int *coloring = new int[numVertices];
    hipMemcpy(coloring, context.coloring, sizeof(int) * numVertices, hipMemcpyDeviceToHost);
    return coloring;
}

int main(int argc, char *argv[]) {
    if(argc < 2) {
        std::cout << "Usage: " << argv[0] << " [in_filename]\n";
        exit(-1);
    }
    const std::string in_filename = argv[1];
    Graph graph = Graph(in_filename);
    struct cudaContext context = setup(graph);

    auto t1 = getTime();
    const int *coloring = jpColoring(context);
    auto t2 = getTime();
    std::cout << "Time: " << getMillis(t1, t2) << "ms\n";

    int numColors = numColorsUsed(coloring, graph.getNumVertices());
    std::cout << "Colors used: " << numColors << "\n";

    bool validColoring = checkColoring(graph, coloring);
    if(!validColoring) {
        std::cout << "Invalid coloring\n";
    }

    freeCudaContext(context);
    delete coloring;
    return 0;
}
