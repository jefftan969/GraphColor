#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <omp.h>

#include "graph.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 512

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if(abort) {
            exit(code);
        }
    }
}
#else
#define cudaCheckError(ans) ans
#endif

// Stores all data structures allocated on CUDA device
struct cudaContext {
    int numVertices;
    int numEdges;
    unsigned int *weights;    // Length numVertices   - List of weights for vertices
    int *vertices;            // Length numVertices+1 - List of graph vertices, in CSR representation
    int *neighbors;           // Length 2*numEdges    - List of vertex neighbors, in CSR representation
    int *worklist;            // Length numVertices   - Boolean array indicating whether each vertex is in worklist
    int *coloring;            // Length numVertices   - Integer array indicating color of each vertex
    hiprandState_t *states;    // Length numVertices   - Random state for each vertex
    int *worklistEmptyFlag;   // Boolean flag indicating whether the worklist is empty
};

/**
 * @brief Set all of the random states to be used for hiprand
 * @param[in] context All data structures allocated on CUDA device
 * @param[in] seed The seed to be used for creating the hiprand states
 */
__global__ void kernelRandInit(struct cudaContext context, unsigned int seed) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if(v < context.numVertices) {
        hiprand_init(seed, v, 0, &context.states[v]);
    }
}

/**
 * @brief Set all vertex weights to be random numbers, which will allow us to find independent sets
 * @param[in] context All data structures allocated on CUDA device
 */
__global__ void kernelSetWeights(struct cudaContext context) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    
    if (v < context.numVertices && context.worklist[v]) {
        context.weights[v] = hiprand(&context.states[v]);
    }
}

/**
 * @brief Creates an independent set of vertices and colors them, and removes colored vertices from worklist
 * @param[in] context All data structures allocated on CUDA device
 * @param[in] color The color to fill in the independent set with
 */
__global__ void kernelColorJP(struct cudaContext context, int color) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int numVertices = context.numVertices;
    const int *vertices = context.vertices;
    const int *neighbors = context.neighbors;
    unsigned int *weights = context.weights;
    int *worklist = context.worklist;
    int *coloring = context.coloring;
    int *worklistEmptyFlag = context.worklistEmptyFlag;

    if(v < numVertices && worklist[v]) {
        // Check neighbors and determine the remaining worklist
        for(int i = vertices[v]; i < vertices[v+1]; i++) {
            int w = neighbors[i];
            if(weights[v] < weights[w]) {
                // Indicate that worklist is not empty and elements still remain
                *worklistEmptyFlag = 0;
                return;
            }
        }
        coloring[v] = color;
        worklist[v] = 0;
    }
}

/**
 * @brief Copy graph representation to CUDA device and allocate all necessary data structures
 * @param[in] graph The graph to color
 * @return context A cudaContext struct containing all device data structures
 */
struct cudaContext setup(const Graph &graph) {
    int numVertices = graph.getNumVertices();
    int numEdges = graph.getNumEdges();
    const int *vertices = graph.getCSRVertices();
    const int *neighbors = graph.getCSRNeighbors();

    // Initialize data structures
    struct cudaContext context;
    context.numVertices = numVertices;
    context.numEdges = numEdges;
    hipMalloc(&context.weights, sizeof(unsigned int) * numVertices);
    hipMalloc(&context.vertices, sizeof(int) * (numVertices + 1));
    hipMalloc(&context.neighbors, sizeof(int) * 2 * numEdges);
    hipMalloc(&context.worklist, sizeof(int) * numVertices);
    hipMalloc(&context.coloring, sizeof(int) * numVertices);
    hipMalloc(&context.states, sizeof(hiprandState_t) * numVertices);
    hipMalloc(&context.worklistEmptyFlag, sizeof(int));
   
    // Note that hipMemset sets values per byte, so 0x01 => 0x01010101 = 16843009
    hipMemcpy(context.vertices, vertices, sizeof(int) * (numVertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(context.neighbors, neighbors, sizeof(int) * 2 * numEdges, hipMemcpyHostToDevice);
    hipMemset(context.worklist, 0x01, sizeof(int) * numVertices);
    hipMemset(context.weights, 0x00, sizeof(int) * numVertices);
    hipMemset(context.coloring, 0x00, sizeof(int) * numVertices);
    hipMemset(context.states, 0x00, sizeof(hiprandState_t) * numVertices);
    hipMemset(context.worklistEmptyFlag, 0x00, sizeof(int));

    return context;
}

/**
 * @brief Frees all data structures in the given CUDA context
 * @param[in] context The CUDA context to free
 */
void freeCudaContext(struct cudaContext context) {
    hipFree(context.weights);
    hipFree(context.vertices);
    hipFree(context.neighbors);
    hipFree(context.worklist);
    hipFree(context.coloring);
    hipFree(context.states);
    hipFree(context.worklistEmptyFlag);
}

/**
 * @brief Performs graph coloring using the JP algorithm
 * @param[in] context A cudaContext sturct containing all device data structures
 * @return coloring The outputted coloring
 */
const int *jpColoring(struct cudaContext context) {
    int numVertices = context.numVertices;

    // Define kernel size and device flags
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((numVertices + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int worklistEmptyFlag = 0;

    // Initialize random state per vertex
    kernelRandInit<<<gridDim, blockDim>>>(context, time(NULL));

    // Loop until worklist is empty
    for(int color = 0; !worklistEmptyFlag; color++) {
        std::cout << "Loop color " << color << "\n";

        // Set random vertex weights for each vertex in worklist
        // All other vertex weights are set beforehand to 0
        hipMemset(context.weights, 0x00, sizeof(unsigned int) * numVertices);
        kernelSetWeights<<<gridDim, blockDim>>>(context);
        hipDeviceSynchronize();
 
        // Resolve conflicts and determine the remaining worklist
        hipMemset(context.worklistEmptyFlag, 0x01, sizeof(int));
        kernelColorJP<<<gridDim, blockDim>>>(context, color);
        hipDeviceSynchronize();
        hipMemcpy(&worklistEmptyFlag, context.worklistEmptyFlag, sizeof(int), hipMemcpyDeviceToHost);
    }
    
    // Retrieve coloring from device
    int *coloring = new int[numVertices];
    hipMemcpy(coloring, context.coloring, sizeof(int) * numVertices, hipMemcpyDeviceToHost);
    return coloring;
}

int main(int argc, char *argv[]) {
    if(argc < 2) {
        std::cout << "Usage: " << argv[0] << " [in_filename]\n";
        exit(-1);
    }
    const std::string in_filename = argv[1];
    Graph graph = Graph(in_filename);
    struct cudaContext context = setup(graph);

    auto t1 = getTime();
    const int *coloring = jpColoring(context);
    auto t2 = getTime();
    std::cout << "Time: " << getMillis(t1, t2) << "ms\n";

    int numColors = numColorsUsed(coloring, graph.getNumVertices());
    std::cout << "Colors used: " << numColors << "\n";

    bool validColoring = checkColoring(graph, coloring);
    if(!validColoring) {
        std::cout << "Invalid coloring\n";
    }

    freeCudaContext(context);
    delete coloring;
    return 0;
}
