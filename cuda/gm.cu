#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <omp.h>

#include "graph.h"

// NOTE: This code currently infinite-loops if the number of colors is greater than COLOR_MASK_SIZE.
// These values are chosen due to the limited number of registers available in CUDA
#define COLOR_MASK_SIZE 500
#define BLOCK_SIZE 128

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if(abort) {
            exit(code);
        }
    }
}
#else
#define cudaCheckError(ans) ans
#endif

// Stores all data structures allocated on CUDA device
struct cudaContext {
    int numVertices;
    int numEdges;
    int blockSize;
    int *vertices;  // Length numVertices+1 - List of graph vertices, in CSR representation
    int *neighbors; // Length 2*numEdges    - List of vertex neighbors, in CSR representation
    int *worklist;  // Length numVertices   - Boolean array indicating whether each vertex is in worklist
    int *coloring;  // Length numVertices   - Integer array indicating color of each vertex
    int *worklistEmptyFlag; // Boolean flag indicating whether the worklist is empty
};

/**
 * @brief Speculatively color vertices in parallel, allowing conflicts which will be resolved later
 * @param[in] context All data structures allocated on CUDA device
 */
__global__ void kernelSpeculate(struct cudaContext context) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int colorMask[COLOR_MASK_SIZE];

    int numVertices = context.numVertices;
    const int *vertices = context.vertices;
    const int *neighbors = context.neighbors;
    int *worklist = context.worklist;
    int *coloring = context.coloring;

    if(v < numVertices && worklist[v]) {
        // Determine colors of all vertex neighbors
        for(int i = vertices[v]; i < vertices[v+1]; i++) {
            int w = neighbors[i];
            colorMask[coloring[w]] = v;
        }
        // Color each vertex with the minimal acceptable color
        for(int i = 0; i < COLOR_MASK_SIZE; i++) {
            if(colorMask[i] != v) {
                coloring[v] = i;
                break;
            }
        }
    }
}

/**
 * @brief Detect conflicts from speculative coloring phase, and remove valid vertices from worklist
 * @param[in] context All data structures allocated on CUDA device
 */
__global__ void kernelResolveConflicts(struct cudaContext context) {
    int v = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int numVertices = context.numVertices;
    const int *vertices = context.vertices;
    const int *neighbors = context.neighbors;
    int *worklist = context.worklist;
    int *coloring = context.coloring;
    int *worklistEmptyFlag = context.worklistEmptyFlag;

    if(v < numVertices && worklist[v]) {
        // Resolve conflicts between neighbors and determine the remaining worklist
        for(int i = vertices[v]; i < vertices[v+1]; i++) {
            int w = neighbors[i];
            if((v < w) && (coloring[v] == coloring[w])) {
                // Indicate that worklist is not empty and elements still remain
                *worklistEmptyFlag = 0;
                return;
            }
        }
        worklist[v] = 0;
    }
}

/**
 * @brief Copy graph representation to CUDA device and allocate all necessary data structures
 * @param[in] graph The graph to color
 * @return context A cudaContext struct containing all device data structures
 */
struct cudaContext setup(const Graph &graph) {
    int numVertices = graph.getNumVertices();
    int numEdges = graph.getNumEdges();
    const int *vertices = graph.getCSRVertices();
    const int *neighbors = graph.getCSRNeighbors();

    // Initialize data structures
    struct cudaContext context;
    context.numVertices = numVertices;
    context.numEdges = numEdges;
    hipMalloc(&context.vertices, sizeof(int) * (numVertices + 1));
    hipMalloc(&context.neighbors, sizeof(int) * 2 * numEdges);
    hipMalloc(&context.worklist, sizeof(int) * numVertices);
    hipMalloc(&context.coloring, sizeof(int) * numVertices);
    hipMalloc(&context.worklistEmptyFlag, sizeof(int));
   
    // Note that hipMemset sets values per byte, so 0x01 => 0x01010101 = 16843009
    hipMemcpy(context.vertices, vertices, sizeof(int) * (numVertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(context.neighbors, neighbors, sizeof(int) * 2 * numEdges, hipMemcpyHostToDevice);
    hipMemset(context.worklist, 0x01, sizeof(int) * numVertices);
    hipMemset(context.coloring, 0x00, sizeof(int) * numVertices);
    hipMemset(context.worklistEmptyFlag, 0x00, sizeof(int));

    return context;
}

/**
 * @brief Frees all data structures in the given CUDA context
 * @param[in] context The CUDA context to free
 */
void freeCudaContext(struct cudaContext context) {
    hipFree(context.vertices);
    hipFree(context.neighbors);
    hipFree(context.worklist);
    hipFree(context.coloring);
    hipFree(context.worklistEmptyFlag);
}

/**
 * @brief Performs graph coloring using the GM algorithm
 * @param[in] context A cudaContext sturct containing all device data structures
 * @return coloring The outputted coloring
 */
const int *gmColoring(struct cudaContext context) {
    int numVertices = context.numVertices;

    // Define kernel size and device flags
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((numVertices + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int worklistEmptyFlag = 0;

    // Loop until worklist is empty
    while(!worklistEmptyFlag) {
        // Determine which colors are permissible for each vertex
        kernelSpeculate<<<gridDim, blockDim>>>(context);
        hipDeviceSynchronize();

        // Resolve conflicts and determine the remaining worklist
        hipMemset(context.worklistEmptyFlag, 0x01, sizeof(int));
        kernelResolveConflicts<<<gridDim, blockDim>>>(context);
        hipMemcpy(&worklistEmptyFlag, context.worklistEmptyFlag, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    
    // Retrieve coloring from device
    int *coloring = new int[numVertices];
    hipMemcpy(coloring, context.coloring, sizeof(int) * numVertices, hipMemcpyDeviceToHost);
    return coloring;
}

int main(int argc, char *argv[]) {
    if(argc < 2) {
        std::cout << "Usage: " << argv[0] << " [in_filename]\n";
        exit(-1);
    }
    const std::string in_filename = argv[1];
    Graph graph = Graph(in_filename);
    struct cudaContext context = setup(graph);

    auto t1 = getTime();
    const int *coloring = gmColoring(context);
    auto t2 = getTime();
    std::cout << "Time: " << getMillis(t1, t2) << "ms\n";

    int numColors = numColorsUsed(coloring, graph.getNumVertices());
    std::cout << "Colors used: " << numColors << "\n";

    bool validColoring = checkColoring(graph, coloring);
    if(!validColoring) {
        std::cout << "Invalid coloring\n";
    }

    freeCudaContext(context);
    delete coloring;
    return 0;
}
